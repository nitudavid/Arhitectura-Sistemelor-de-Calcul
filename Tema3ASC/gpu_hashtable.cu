#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <ctime>
#include <sstream>
#include <string>
#include "test_map.hpp"
#include "gpu_hashtable.hpp"

using namespace std;

/*
Allocate CUDA memory only through glbGpuAllocator
hipMalloc -> glbGpuAllocator->_cudaMalloc
hipMallocManaged -> glbGpuAllocator->_cudaMallocManaged
hipFree -> glbGpuAllocator->_cudaFree
*/

/******** Kernel Functions ********/

//Function that performs reshape redistributing the elements of a hashTable into a new hashTable with a different size
__global__ void reshapeKernel(Probe* oldhashTable, Probe* newhashTable, int oldSize, int newSize) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < oldSize) {
        if (oldhashTable[idx].key != KEY_INVALID) {
            uint32_t key = oldhashTable[idx].key;
			//Hash function
            uint32_t hash = key % newSize;

            //Perform linear probing to find an empty slot
            while (atomicCAS((unsigned int*)&newhashTable[hash].key, KEY_INVALID, key) != KEY_INVALID) {
                hash = (hash + 1) % newSize;
            }

            //Copy the value to the new hash table
            atomicExch((unsigned int*)&newhashTable[hash].value, oldhashTable[idx].value);
        }
    }
}

//Function that inserts a batch of key and value into a hashTable
__global__ void insertBatchKernel(Probe* hashTable, int* keys, int* values, int numKeys, int size, int* tempSize) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numKeys) {
        int key = keys[idx];
        int value = values[idx];
		//Hash function
        int hash = key % size;

        //Perform linear probing to find an empty slot or the matching key
        while (atomicCAS((unsigned int*)&hashTable[hash].key, 0, key) != 0 && hashTable[hash].key != key) {
            hash = (hash + 1) % size;
        }

        //Update the value when we find an empty slot or the matching key
        atomicExch(&hashTable[hash].value, value);

        //Increment the current size if its a new key
        if (hashTable[hash].key == key) {
            atomicAdd(tempSize, 1);
        }
    }
}

//Function that returns a batch of values from a hashTable
__global__ void getBatchKernel(Probe* hashTable, int* keys, int* values, int numKeys, int size) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numKeys) {
        int key = keys[idx];
		//Hash function
        int hash = key % size;

        //Perform linear probing to find the matching key or an empty slot
        while (hashTable[hash].key != key && hashTable[hash].key != 0) {
            hash = (hash + 1) % size;
        }

        //If the key is found, retrieve the value
        if (hashTable[hash].key == key) {
            atomicExch(&(values[idx]), hashTable[hash].value);
        }
    }
}



/**
 * Function constructor GpuHashTable
 * Performs init
 * Example on using wrapper allocators _cudaMalloc and _cudaFree
 */
GpuHashTable::GpuHashTable(int size) 
	: totalSize(size), tempSize(0)
{
	Probe* hashTable;
	glbGpuAllocator->_cudaMallocManaged((void**)&hashTable, (int)(size * sizeof(Probe)));
	hipMemset((void*)hashTable, 0, (int)(size * sizeof(Probe)));
	this->hashTable = hashTable;
}

/**
 * Function desctructor GpuHashTable
 */
GpuHashTable::~GpuHashTable() {
	glbGpuAllocator->_cudaFree((void*)hashTable);
}

/**
 * Function reshape
 * Performs resize of the hashtable based on load factor
 */
void GpuHashTable::reshape(int numBucketsReshape) {

	//Creating the new hashTable, allocating memory and initializing it with 0
	Probe* newHashTable;
	glbGpuAllocator->_cudaMallocManaged((void**)&newHashTable, (int)(numBucketsReshape * sizeof(Probe)));
	hipMemset((void*)newHashTable, 0, (int)(numBucketsReshape * sizeof(Probe)));

    //Calculate the number of blocks based on the number of keys and available threads
	int blocks, threads;
    hipDeviceProp_t DeviceProp;
    hipGetDeviceProperties(&DeviceProp, 0);
    threads = DeviceProp.maxThreadsPerBlock;
    if (totalSize % threads == 0) {
        blocks = totalSize / threads;
    }
    else {
        blocks = totalSize / threads + 1;
    }

	//Transfering the old hashTable to the new hashTable using the reshape function
	reshapeKernel<<<blocks, threads>>>(hashTable, newHashTable, totalSize, numBucketsReshape);
	//Ensuring that all processes are completed before the next thread continues executing the next operation
	hipDeviceSynchronize();

	//Free-ing the old hashTable and reseting the variables
	glbGpuAllocator->_cudaFree((void*)hashTable);
	hashTable = newHashTable;
	totalSize = numBucketsReshape;
}

/**
 * Function insertBatch
 * Inserts a batch of key:value, using GPU and wrapper allocators
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	
	//Initializing the new keys and values, allocating memory for them, and copy-ing them from the Host to the Device
	int* newKeys = 0;
	int* newValues = 0;
	glbGpuAllocator->_cudaMalloc((void**)&newKeys, (int)(numKeys * sizeof(int)));
	glbGpuAllocator->_cudaMalloc((void**)&newValues, (int)(numKeys * sizeof(int)));
	hipMemcpy(newKeys, keys, (int)(numKeys * sizeof(int)), hipMemcpyHostToDevice);
	hipMemcpy(newValues, values, (int)(numKeys * sizeof(int)), hipMemcpyHostToDevice);

	//Seeing if the hashTable needs resizing based on its comparison to the loadFactor
	if (tempSize + numKeys > totalSize * 0.9) {
		reshape(totalSize * 2);
	}

	//Copy-ing the new size(in case it changed, else it is the same one but we still copy it) to the Device
	int* newtempSize = 0;
	glbGpuAllocator->_cudaMalloc((void**)&newtempSize, sizeof(int));
	hipMemcpy(newtempSize, &tempSize, sizeof(int), hipMemcpyHostToDevice);

	//Calculate the number of blocks based on the number of keys and available threads
	int blocks, threads;
    hipDeviceProp_t DeviceProp;
    hipGetDeviceProperties(&DeviceProp, 0);
    threads = DeviceProp.maxThreadsPerBlock;
    if (numKeys % threads == 0) {
        blocks = numKeys / threads;
    }
    else {
        blocks = numKeys / threads + 1;
    }

	//Inserting the batch
	insertBatchKernel<<<blocks, threads>>>(hashTable, newKeys, newValues, numKeys, totalSize, newtempSize);
	//Ensuring that all processes are completed before the next thread continues executing the next operation
	hipDeviceSynchronize();

	//Copy-ing the size after its change to the Host
	hipMemcpy(&tempSize, newtempSize, sizeof(int), hipMemcpyDeviceToHost);

	//Free-ing the variables used for insertBatch
	glbGpuAllocator->_cudaFree(newKeys);
	glbGpuAllocator->_cudaFree(newValues);
	glbGpuAllocator->_cudaFree(newtempSize);

	return true;
}

/**
 * Function getBatch
 * Gets a batch of key:value, using GPU
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {

	//Initializing the new keys and values, allocating memory for them, and copy-ing them from the Host to the Device
	int* newKeys;
	int* newValues;
	glbGpuAllocator->_cudaMalloc((void**)&newKeys, (int)(numKeys * sizeof(int)));
	glbGpuAllocator->_cudaMalloc((void**)&newValues, (int)(numKeys * sizeof(int)));
	hipMemcpy(newKeys, keys, (int)(numKeys * sizeof(int)), hipMemcpyHostToDevice);
	
	//Calculate the number of blocks based on the number of keys and available threads
	int blocks, threads;
    hipDeviceProp_t DeviceProp;
    hipGetDeviceProperties(&DeviceProp, 0);
    threads = DeviceProp.maxThreadsPerBlock;
    if (numKeys % threads == 0) {
        blocks = numKeys / threads;
    }
    else {
        blocks = numKeys / threads + 1;
    }

	//Returning the batch
	getBatchKernel<<<blocks, threads>>>(hashTable, newKeys, newValues, numKeys, totalSize);
	//Ensuring that all processes are completed before the next thread continues executing the next operation
	hipDeviceSynchronize();

	//Copy-ing the new values to the host and storing them so we can return them 
	int* result = (int*)malloc((int)(numKeys * sizeof(int)));
	hipMemcpy(result, newValues, (int)(numKeys * sizeof(int)), hipMemcpyDeviceToHost);

	//Free-ing the variables used for getBatch
	glbGpuAllocator->_cudaFree(newKeys);
	glbGpuAllocator->_cudaFree(newValues);

	return result;
}
